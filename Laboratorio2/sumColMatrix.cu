#include <hip/hip_runtime.h>

#include <iostream>

using namespace std;
const int DIMBLOCKX=32;
//DEVICE


__global__ void kernelSum_Column_Matrix(float* matrix, float* array, int tam){
    __shared__ float shareMatrix[DIMBLOCKX];

    float value=0;
    int col=blockIdx.x;
    int step= tam/blockDim.x;
    int posIni= col*tam+threadIdx.x*step;
    for(int i=0;i<step;i++){
        value=value+matrix[posIni+i];
    }
    
    shareMatrix[threadIdx.x]=value;
    __syncthreads();

    if(threadIdx.x==0){
        for(int j=1;j<blockDim.x;j++){
            shareMatrix[0]=shareMatrix[0]+shareMatrix[j];
        }
        array[blockIdx.x]=shareMatrix[0];
    }
}

//HOST
int main(){
    int row=512;   
    int col=512;

    float* matrix= (float*) malloc(sizeof(float)*row*col);
    float* matrix_DEVICE= NULL;
    float* array_DEVICE= NULL;

    float* array=new float[col];
    for(int i=0;i<row;i++){
        for(int j=0; j<col;j++){
            matrix[i*col+j]=j;
        }
    }

    hipMalloc((void**)&matrix_DEVICE,sizeof(float)*row*col);
    hipMalloc((void**)&array_DEVICE, col*sizeof(float));

    hipMemcpy(matrix_DEVICE,matrix,sizeof(float)*row*col,hipMemcpyHostToDevice);
    dim3 dimGrid(col,1);
    dim3 dimBlock(row/DIMBLOCKX,1);
    
    kernelSum_Column_Matrix<<< dimGrid , dimBlock >>>(matrix_DEVICE,array_DEVICE,col);
    
    hipMemcpy(array,array_DEVICE,sizeof(float)*col,hipMemcpyDeviceToHost);
    for( int index = 0; index<col ; index++){
		cout<<array[index]<<"  ";
    }
    
    hipFree(matrix_DEVICE);
    hipFree(array_DEVICE);

    delete[] array;
    delete[] matrix;
    
}