#include <hip/hip_runtime.h>

#include <iostream>

using namespace std;

//DEVICE

__global__ void kernelVector_suma_constante(float* array, int _size, int _constant){
    int idx= blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < _size){
        array[idx] = array[idx]+_constant;
    }
}


//HOST
int main(){
    int size = 1000000;
    float* arr = new float[size];
    float* arr_DEVICE= NULL;

    for (int index = 0; index < size; index++){
        arr[index] = index;
    }
    
    hipMalloc((void**)&arr_DEVICE,size * sizeof(float));
    hipMemcpy(arr_DEVICE, arr,size * sizeof(float), hipMemcpyHostToDevice);

    kernelVector_suma_constante <<< ceil(size/512),512>>>(arr_DEVICE,size,65);
    
    hipMemcpy(arr,arr_DEVICE,size * sizeof (float), hipMemcpyDeviceToHost);
    for ( int index = 0; index<100; index++){
        cout<<arr[index]<<endl;
    }
    
    hipFree(arr_DEVICE);
    delete[] arr;
}