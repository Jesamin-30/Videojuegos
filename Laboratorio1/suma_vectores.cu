#include <hip/hip_runtime.h>

#include <iostream>

using namespace std;

//DEVICE

__global__ void kernelSuma_Vectores(float* array_A, float* array_B, int _size){
    int idx= blockIdx.x*blockDim.x+threadIdx.x;
    if(idx<_size){
        array_A[idx] = array_A[idx] + array_B[idx];
    }
}


//HOST
int main(){
    int size= 1000000;
    float* array_A= new float[size];
    float* array_B= new float[size];

    float* array_A_DEVICE=NULL;
    float* array_B_DEVICE=NULL;

    for (int index = 0; index < size ; index++){
        array_A[index]= index;
        array_B[index]= index;
    }

    hipMalloc((void**)&array_A_DEVICE,size*sizeof(float));
    hipMalloc((void**)&array_B_DEVICE,size*sizeof(float));

    hipMemcpy(array_A_DEVICE,array_A,size*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(array_B_DEVICE,array_B,size*sizeof(float),hipMemcpyHostToDevice);

    kernelSuma_Vectores<<<ceil(size/512),512>>>(array_A_DEVICE,array_B_DEVICE,size);
    
    hipMemcpy(array_A,array_A_DEVICE,size*sizeof(float),hipMemcpyDeviceToHost);
    for( int index=0 ; index< 100 ; index++){
        cout<<array_A[index]<< endl;
    }
    hipFree(array_A_DEVICE);
    hipFree(array_B_DEVICE);

    delete[] array_A;
    delete[] array_B;
}
